#include "hip/hip_runtime.h"
#include<opencv2/opencv.hpp>
#include<iostream>
#include <thread>
// #include "circuqueue.hpp"
#include "hip/hip_runtime.h"
#include ""
static const int INPUT_H = 640;
static const int INPUT_W = 640;

static float data[3 * INPUT_H * INPUT_W];

__global__ void convert(uchar *mat ,float* data){

    
    uint ix=threadIdx.x+blockDim.x*blockIdx.x;
    uint iy=threadIdx.y+blockDim.y*blockIdx.y;

    int i=ix+iy*(gridDim.x*blockDim.x);

    data[i] = mat[i+2]/ 255.0;
    data[i + INPUT_H * INPUT_W] = mat[i+1] / 255.0;
    data[i + 2 * INPUT_H * INPUT_W] = mat[i+0] / 255.0;
    

}
void gpu_convert(uchar *d_src ,float*d_dst,cv::Mat &frame){
	hipMemcpy(d_src,frame.data,3 * INPUT_H * INPUT_W,hipMemcpyHostToDevice);
	dim3 grid1(20,20);
	dim3 block(32,32);
	convert<<<grid1,block>>>(d_src,d_dst);
	hipDeviceSynchronize();
}
void cpu_convert(cv::Mat &pr_img){

	for (int i = 0; i < INPUT_H * INPUT_W; i++) {
		data[i] = pr_img.at<cv::Vec3b>(i)[2] / 255.0;
		data[i + INPUT_H * INPUT_W] = pr_img.at<cv::Vec3b>(i)[1] / 255.0;
		data[i + 2 * INPUT_H * INPUT_W] = pr_img.at<cv::Vec3b>(i)[0] / 255.0;
	}
}

// int main(int argc, char** argv) {
  
// 	// 打开文件
// 	cv::VideoCapture capture;
// 	capture.open("/home/xu/视频/01.avi");
// 	if (!capture.isOpened()) {
// 		printf("could not read this video file...\n");
// 		return -1;
// 	}
//     cv::Mat frame;
// 	// cv::cuda::GpuMat dst,src;
// 	// src.upload(frame);
// 	unsigned char *array=new unsigned char[3 * INPUT_H * INPUT_W];
	
// 	uchar* d_src = NULL;
//     // uchar3* d_dst = NULL;
// 	uchar *d_dst=NULL;
//     hipMalloc((void**)&d_src,3 * INPUT_H * INPUT_W);
// 	hipMalloc((void**)&d_dst,3 * INPUT_H * INPUT_W);

//     auto start = std::chrono::system_clock::now();
//     while (true) {

// 		capture>>frame;
// 		if (frame.empty())
// 		{
// 			break;
// 		}
// 		cv::imshow("01",frame);
// 		cv::waitKey(1);
		
// 		// gpu_convert(d_src ,d_dst,frame);
//         cpu_convert(frame);
//     }
// 	auto end = std::chrono::system_clock::now();
//     std::cout <<"time: "<<std::setw(2)<< std::chrono::duration_cast<std::chrono::milliseconds>(end-start).count() << "ms  " ;
// }